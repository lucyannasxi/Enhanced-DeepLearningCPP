#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace graphdl
{
namespace core
{
namespace cuda
{
__global__ void setupKernel(hiprandState* state, size_t seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}

__global__ void uniformRandomKernel(hiprandState* state, float* memory,
                                    size_t size, float min, fl