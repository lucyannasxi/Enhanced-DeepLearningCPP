#include "hip/hip_runtime.h"
#include "weights.h"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace graphdl
{
namespace core
{
namespace cuda
{
__global__ void setup_kernel(hiprandState* state)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void initWeightsKernel(hiprandState* state, size_t size,
                                  float* output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) output[id] = hiprand_uniform(state + id) * 2. - 1.;
}
